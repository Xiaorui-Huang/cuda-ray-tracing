#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    hipDeviceProp_t prop;
    int device;

    // Get the device id of the currently active GPU.
    hipGetDevice(&device);

    // Get the properties of the device.
    hipGetDeviceProperties(&prop, device);

    // Print out the size of the constant and shared memory.
    std::cout << "Total constant memory: " << prop.totalConstMem << " bytes" << std::endl;
    std::cout << "Shared memory per block: " << prop.sharedMemPerBlock << " bytes" << std::endl;

    return 0;
}
