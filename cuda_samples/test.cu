
#include <hip/hip_runtime.h>
#include <cstdio>
#include <tuple>

using Point = std::tuple<float, float, float>;

__global__ void addPoints(const Point *a, const Point *b, Point *result) {
    int idx = threadIdx.x;
    if (idx == 0) {
        std::get<0>(*result) = std::get<0>(*a) + std::get<0>(*b);
        std::get<1>(*result) = std::get<1>(*a) + std::get<1>(*b);
        std::get<2>(*result) = std::get<2>(*a) + std::get<2>(*b);
    }
}

int main() {
    Point h_a(1.0f, 2.0f, 3.0f);
    Point h_b(4.0f, 5.0f, 6.0f);
    Point h_result;

    Point *d_a, *d_b, *d_result;

    hipMalloc(&d_a, sizeof(Point));
    hipMalloc(&d_b, sizeof(Point));
    hipMalloc(&d_result, sizeof(Point));

    hipMemcpy(d_a, &h_a, sizeof(Point), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &h_b, sizeof(Point), hipMemcpyHostToDevice);

    addPoints<<<1, 1>>>(d_a, d_b, d_result);

    hipMemcpy(&h_result, d_result, sizeof(Point), hipMemcpyDeviceToHost);

    printf("Result: (%f, %f, %f)\n", std::get<0>(h_result), std::get<1>(h_result), std::get<2>(h_result));

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);

    return 0;
}
