#include <hip/hip_runtime.h>
#include <iostream>

#define N 256 // example matrix size

// Kernel to add two matrices
__global__ void MatAdd(float MatA[N][N], float MatB[N][N], float MatC[N][N]) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < N && j < N)
        MatC[i][j] = MatA[i][j] + MatB[i][j];
}

int main() {
    float MatA[N][N], MatB[N][N], MatC[N][N];

    // Initializing matrices with some values for testing
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++) {
            MatA[i][j] = i + j;
            MatB[i][j] = i - j;
        }

    float(*d_MatA)[N], (*d_MatB)[N], (*d_MatC)[N];

    // Allocate memory on the device
    hipMalloc((void **)&d_MatA, N * N * sizeof(float));
    hipMalloc((void **)&d_MatB, N * N * sizeof(float));
    hipMalloc((void **)&d_MatC, N * N * sizeof(float));

    // Copy matrices from host to device
    hipMemcpy(d_MatA, MatA, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_MatB, MatB, N * N * sizeof(float), hipMemcpyHostToDevice);

    // Kernel launch
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (N + threadsPerBlock.y - 1) / threadsPerBlock.y);
    MatAdd<<<numBlocks, threadsPerBlock>>>(d_MatA, d_MatB, d_MatC);

    // Copy result matrix from device to host
    hipMemcpy(MatC, d_MatC, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // (Optional) Print the result
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++)
            std::cout << MatC[i][j] << " ";
        std::cout << std::endl;
    }

    // Cleanup and free memory
    hipFree(d_MatA);
    hipFree(d_MatB);
    hipFree(d_MatC);

    return 0;
}
